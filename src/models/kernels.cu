#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

namespace Generators {
namespace cuda {

__global__ void Gpt_UpdatePositionIds(int32_t* positions, int batch_beam_size, int current_length) {
  for (int i = 0; i < batch_beam_size; i++)
    positions[i] = current_length - 1;
}

void LaunchGpt_UpdatePositionIds(int32_t* positions, int batch_beam_size, int current_length, hipStream_t stream) {
  Gpt_UpdatePositionIds<<<1, 1, 0, stream>>>(positions, batch_beam_size, current_length);
}

__global__ void Gpt_UpdateMask(int32_t* mask_data, const int32_t* old_mask_data, int batch_beam_size, int current_length) {
  for (int i = 0; i < batch_beam_size; i++) {
    for (int j = 0; j < current_length - 1; j++) {
      mask_data[i * current_length + j] = old_mask_data[i * (current_length - 1) + j];
    }
    mask_data[i * current_length + current_length - 1] = 1;
  }
}

void LaunchGpt_UpdateMask(int32_t* mask_data, const int32_t* old_mask_data, int batch_beam_size, int current_length, hipStream_t stream) {
  Gpt_UpdateMask<<<1, 1, 0, stream>>>(mask_data, old_mask_data, batch_beam_size, current_length);
}

__global__ void Gpt_UpdatePositionIds(int64_t* positions, int batch_beam_size, int current_length) {
  for (int i = 0; i < batch_beam_size; i++) {
    positions[i] = current_length - 1;
  }
}

void LaunchGpt_UpdatePositionIds(int64_t* positions, int batch_beam_size, int current_length, hipStream_t stream) {
  Gpt_UpdatePositionIds<<<1, 1, 0, stream>>>(positions, batch_beam_size, current_length);
}

__global__ void Gpt_UpdateMask(int64_t* mask_data, const int64_t* old_mask_data, int batch_beam_size, int current_length) {
  for (int i = 0; i < batch_beam_size; i++) {
    for (int j = 0; j < current_length - 1; j++) {
      mask_data[i * current_length + j] = old_mask_data[i * (current_length - 1) + j];
    }
    mask_data[i * current_length + current_length - 1] = 1;
  }
}

void LaunchGpt_UpdateMask(int64_t* mask_data, const int64_t* old_mask_data, int batch_beam_size, int current_length, hipStream_t stream) {
  Gpt_UpdateMask<<<1, 1, 0, stream>>>(mask_data, old_mask_data, batch_beam_size, current_length);
}

__global__ void ConvertFp16ToFp32(const half* src, float* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    dst[idx] = __half2float(src[idx]);
  }
}

void LaunchFp16ToFp32(const uint16_t* fp16, float* fp32, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp16ToFp32<<<num_blocks, block_size, 0, stream>>>(reinterpret_cast<const half*>(fp16), fp32, count);
}

__global__ void ConvertInt32ToInt64(const int32_t* src, int64_t* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    dst[idx] = src[idx];
  }
}

void LaunchInt32ToInt64(const int32_t* src, int64_t* dst, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertInt32ToInt64<<<num_blocks, block_size, 0, stream>>>(src, dst, count);
}

}  // namespace cuda
}  // namespace Generators
